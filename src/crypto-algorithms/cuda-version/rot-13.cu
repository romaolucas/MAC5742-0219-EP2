#include "hip/hip_runtime.h"
//standard includes
#include <stdio.h>
#include <stdlib.h>

//stat struct
#include <sys/stat.h>


extern "C" {
//read file and size functions
    #include "util.h"
}

#define N (2048 * 2048)
#define NUM_THREADS 512



__global__ void rot13(BYTE* str, size_t *len)
{
    int case_type, idx;
    idx = threadIdx.x + blockDim.x * blockIdx.x;
    case_type = 'A';
    if (idx < *len) {
        if (str[idx] < 'A' || (str[idx] > 'Z' && str[idx] < 'a') || str[idx] > 'z') {
            return;
         } else {
            if (str[idx] >= 'a') {
                case_type = 'a';
            }
            str[idx] = (str[idx] + 13) % (case_type + 26);
            if (str[idx] < 26) {
                str[idx] += case_type;
            }
        }
    }
}

int main(int argc, char *argv[])
{
    hipError_t err = hipSuccess;
    BYTE *data;
    BYTE *d_data = NULL;
    BYTE *enc_data;
    size_t len;
    size_t *d_len = NULL;

    if (argc != 3) {
        printf("Uso: ./rot-13 nome_arquivo nome_arquivo_criptografado\n");
        exit(EXIT_FAILURE);
    }

    data = read_file(argv[1]);
    len = get_file_size();
    enc_data = (BYTE *) malloc(len * sizeof(BYTE));

    err = hipMalloc(&d_data, len * sizeof(BYTE));
    if (err != hipSuccess) {
        fprintf(stderr, "Falha na alocacao do d_data\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_len, sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "Falha na alocacao do d_len\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_data, data, len * sizeof(BYTE), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Falha na copia do data para o device\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_len, &len, sizeof(size_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Falha na copia do len para o device\n");
        exit(EXIT_FAILURE);
    }


    rot13 <<<N/NUM_THREADS, NUM_THREADS>>>(d_data, d_len);
    
    err = hipMemcpy(enc_data, d_data, len * sizeof(BYTE), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Falha na copia do data para o host\n");
        exit(EXIT_FAILURE);
    }

    FILE *enc_file = fopen(argv[2], "wb");
    fwrite(enc_data, len * sizeof(BYTE), 1, enc_file);
    free(enc_data);
    fclose(enc_file);
    free(data);
    hipFree(d_data);
    hipFree(d_len);
    return(0);
}
