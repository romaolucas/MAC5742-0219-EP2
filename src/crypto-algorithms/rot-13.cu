#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "util.c"

__global__ void rot13(BYTE* str, int len)
{
   int case_type, idx;
   for (idx = 0; idx < len; idx++) {
      // Only process alphabetic characters.
      if (str[idx] < 'A' || (str[idx] > 'Z' && str[idx] < 'a') || str[idx] > 'z')
         continue;
      // Determine if the char is upper or lower case.
      if (str[idx] >= 'a')
         case_type = 'a';
      else
         case_type = 'A';
      // Rotate the char's value, ensuring it doesn't accidentally "fall off" the end.
      str[idx] = (str[idx] + 13) % (case_type + 26);
      if (str[idx] < 26)
         str[idx] += case_type;
   }
}

int main(int argc, char *argv[])
{
    //printf("ROT-13 tests: %s\n", rot13_test() ? "SUCCEEDED" : "FAILED");
    BYTE *data;
    struct stat st;

    if (argc != 3) {
        printf("Uso: ./rot-13 nome_arquivo nome_arquivo_criptografado\n");
        exit(EXIT_FAILURE);
    }

    data = read_file(argv[1]);
    rot13 <<<1, 1>>>(data, sizeof(BYTE) * st.st_size);
    free(data);
    return(0);
}